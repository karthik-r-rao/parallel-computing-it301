#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int SWEEPS = 100;

void printArray(int* arr, int size) {
	for (int i = 0; i < size; i++) {
		printf("%d ", arr[i]);
	}
	printf("\n");
}

int getSize(FILE* fptr) {
	int size = 0;
	int temp;
	while (fscanf(fptr, "%d", &temp) == 1) {
		size++;
	}
	return size;
}

void getArray(FILE* fptr, int* arr) {
	int i = 0;
	while (fscanf(fptr, "%d", &arr[i]) == 1) {
		i++;
	}
}

// CUDA kernel for performing Ising Annealing in parallel
__global__ void isingAnnealingStep(int* d_flat, int* d_places, int* d_state, int* count, int vertices, int N){
    *count = N;
    int vertex = blockDim.x*blockIdx.x + threadIdx.x;
    if(vertex < vertices){
        int sigmaI = d_state[vertex];
        int H = 0;
        for (int i = d_places[vertex]; i < d_places[vertex + 1]; i += 2) {
            int sigmaJ = d_state[d_flat[i]];
            int J = d_flat[i + 1];
            H -= (J * sigmaI * sigmaJ);
        }
        __syncthreads();
        hiprandState_t random;
        hiprand_init(0, 0, 0, &random);
        int s = H / sigmaI;
        if (s > 0) {
            d_state[vertex] = -1;
        }
        else if (s < 0) {
            d_state[vertex] = 1;
        }
        else {
            d_state[vertex] = 1 - 2 * (hiprand(&random) % 2);
        }
        __syncthreads();
        hiprand_init(0, 0, 0, &random);
        if(*count > 0){
            int index = hiprand(&random) % vertices;
            if (d_state[index] == 1) {
                d_state[index] = -1;
            }
            else {
                d_state[index] = 1;
            }
            *count--;
        } 
    }
}

// utility function to compute Hamiltonian given a vertex
// ***not used in the parallel implementation***
int computeHamiltonian(int* flat, int* places, int* state, int vertex) {
	int sigmaI = state[vertex];
	int H = 0;
	for (int i = places[vertex]; i < places[vertex + 1]; i += 2) {
		int sigmaJ = state[flat[i]];
		int J = flat[i + 1];
		H -= (J * sigmaI * sigmaJ);
	}
	return H;
}


int main(){
	FILE* fptr;

	// load array $flat 
	fptr = fopen("flat.txt", "r");
	if (fptr == NULL)
	{
		printf("Error!");
		return -1;
	}
	// get number of integers for dynamic memory allocation on host
	int fsize = getSize(fptr);
	printf("Number of integers present in flat.txt: %d\n", fsize);

	// read values in text file to array $flat
	int* flat = (int*)malloc(sizeof(int) * fsize);
	fseek(fptr, 0, SEEK_SET);
	getArray(fptr, flat);

	//printArray(flat, fsize);
	fclose(fptr);

	/**********/

	// load array $places
	fptr = fopen("places.txt", "r");
	if (fptr == NULL)
	{
		printf("Error!");
		return -1;
	}
	// get number of integers for dynamic memory allocation on host
	int psize = getSize(fptr);
	printf("Number of integers present in places.txt: %d\n", psize);

	// read values in text file to array $places
	int* places = (int*)malloc(sizeof(int) * psize);
	fseek(fptr, 0, SEEK_SET);
	getArray(fptr, places);

	//printArray(places, psize);
	fclose(fptr);

	/**********/

	int vertices = psize - 1;
	int* state = (int*)malloc(sizeof(int) * vertices);
    printf("Number of vertices: %d\n", vertices);

	// initialize states randomly
	for (int i = 0; i < vertices; ++i) {
		state[i] = 1 - 2 * (rand() % 2);
	}

    double initial_energy = 0;
	for (int i = 0; i < vertices; i++) {
		int H = computeHamiltonian(flat, places, state, i);
		initial_energy += H;
	}
	printf("Initial energy: %f\n", initial_energy / 2);

    // allocate memory on device
    int* d_flat; 
    int* d_places;
    int* d_state;
    int* count;
    hipMalloc((void**)&count, sizeof(int));
    hipMalloc((void**)&d_flat, fsize * sizeof(int));
    hipMalloc((void**)&d_places, psize*sizeof(int));
    hipMalloc((void**)&d_state, vertices*sizeof(int));

    hipMemcpy(d_flat, flat, fsize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_places, places, psize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_state, state, vertices*sizeof(int), hipMemcpyHostToDevice);

    int threads = 1024;
    int blocks = (vertices / 1024) + 1;
    int N = vertices*0.75;
    clock_t begin = clock();
	for (int i = 0; i < SWEEPS; i++) {
        isingAnnealingStep<<<blocks, threads>>>(d_flat, d_places, d_state, count, vertices, N);
        //cudaDeviceSynchronize();
        N *= 0.9;
    }
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    hipMemcpy(flat, d_flat, fsize*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(places, d_places, psize*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(state, d_state, vertices*sizeof(int), hipMemcpyDeviceToHost);

    double final_energy = 0;
	for (int i = 0; i < vertices; i++) {
		int H = computeHamiltonian(flat, places, state, i);
		final_energy += H;
	}
	printf("Final energy: %f\n", final_energy / 2);

    printf("Time taken for parallel Ising Annealing: %f seconds\n", time_spent);

    // store results in txt files
    // $time_spent $initial_energy $final_energy 
    printf("Storing parallel code's results in results.txt...\n");
    fptr = fopen("results.txt", "a");
    fprintf(fptr, "Parallel\t%d\t%f\t%f\t%f\n", vertices, time_spent, initial_energy, final_energy);
    fclose(fptr);
    printf("Finished writing to results.txt\n");

	// free dynamic memory on host and device
	free(flat);
	free(places);
	free(state);
    hipFree(d_flat);
    hipFree(d_places);
    hipFree(d_state);
    
	return 0;
}